#include "hip/hip_runtime.h"
#include "BilateralFilter.cuh"
#include "LowpassFilter.cuh"
#include "Common.h"
#include <Windows.h>
#include <cuda_gl_interop.h>
#include <iostream>
#include <cassert>
#include <vector>

#define PI 3.14159265359f

__constant__ float d_Gaussian[512];

texture<float, hipTextureType2D, hipReadModeElementType> tex_depth_original;
texture<float, hipTextureType2D, hipReadModeElementType> tex_depth;
texture<float, hipTextureType2D, hipReadModeElementType> tex_dogFiltered;
surface<void, cudaSurfaceType2D> surf_outline;

inline __host__ __device__ float euclideanLength(float a, float b, float sigma){
	return expf(-((a - b) * (a - b)) / (2.0f * sigma * sigma));
}

inline __host__ __device__ float gaussian(float x, float y, float sigma){
	return expf(-(x * x + y * y) / (2.0f * sigma * sigma)) / (2.0f * PI * sigma * sigma);
}

inline __host__ __device__ float gaussian(float x, float sigma){
	return expf(-(x * x) / (2.0f * sigma * sigma)) / (sqrtf(2.0f * PI) * sigma);
}

__constant__ int restorePos;
__constant__ float projectionMatrix_3_2, projectionMatrix_2_2;
//__constant__ float invProjectionMatrix[4];
__constant__ float4 invProjectionMatrix[4];

inline __host__ __device__ float4 mat4Mult(float4 *m, float4 v){
	//float4 Mov0 = make_float4(v.x);
	//float4 Mov1 = make_float4(v.y);
	//float4 Mul0 = m[0] * Mov0;
	//float4 Mul1 = m[1] * Mov1;
	//float4 Add0 = Mul0 + Mul1;
	//float4 Mov2 = make_float4(v.z);
	//float4 Mov3 = make_float4(v.w);
	//float4 Mul2 = m[2] * Mov2;
	//float4 Mul3 = m[3] * Mov3;
	//float4 Add1 = Mul2 + Mul3;
	//return Add0 + Add1;

	return m[0] * make_float4(v.x) + m[1] * make_float4(v.y) + m[2] * make_float4(v.z) + m[3] * make_float4(v.w);
}

inline __host__ __device__ float4 backProjection(float P32, float P22, float4 *invP, float4 p_ndc){
	float w_c = P32 / (P22 + p_ndc.z);
	float4 p_clip = w_c * p_ndc;
	return mat4Mult(invP, p_clip);
}

inline __host__ __device__ float backProjection(float P32, float P22, float z_ndc)
{
	return -P32 / (P22 + z_ndc);
}

__global__ void d_GaussianFilter(float *out, int width, int height, int radius, float sigma_s, float sigma_r){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * width;
	if (offset >= width * height){
		return;
	}

	float center = tex2D(tex_depth, x, y);
	if (center == 1.0f){
		out[offset] = 1.0f;
		return;
	}
	float4 center_eye = backProjection(projectionMatrix_3_2, projectionMatrix_2_2, invProjectionMatrix,
										make_float4(((float)x / (float)width) * 2.0f - 1.0f,
													((float)y / (float)height) * 2.0f - 1.0f,
													center * 2.0f - 1.0f,
													1.0f));

	float sum = 0.0f;
	float factor = 0.0f;
	float t = 0.0f;
	for (int i = -radius; i <= radius; ++i){
		for (int j = -radius; j <= radius; ++j){
			float sample = tex2D(tex_depth, x + i, y + j);
			float4 sample_eye = backProjection(projectionMatrix_3_2, projectionMatrix_2_2, invProjectionMatrix,
												make_float4(((float)(x + i) / (float)width) * 2.0f - 1.0f,
															((float)(y + i) / (float)height) * 2.0f - 1.0f,
															sample * 2.0f - 1.0f,
															1.0f));

			//if (restorePos){
				factor = gaussian(center_eye.x - sample_eye.x, center_eye.y - sample_eye.y, sigma_s) * euclideanLength(sample_eye.z, center_eye.z, sigma_r);
			//}
			//else{
			//	factor = gaussian(i, j, sigma_s) * euclideanLength(sample_eye.z, center_eye.z, sigma_r);
			//}

			t += factor * sample;
			sum += factor;
		}
	}

	out[offset] = t / sum;
}
//
//__global__ void ortho(float *out, int width, int height){
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//	int offset = x + y * width;
//	if (offset >= width * height){
//		return;
//	}
//
//	float center = tex2D(tex_depth, x, y);
//	float4 center_eye = backProjection(make_float4(((float)x / (float)width) * 2.0f - 1.0f,
//													((float)y / (float)height) * 2.0f - 1.0f,
//													center * 2.0f - 1.0f,
//													1.0f));
//
//	float4 frustum = backProjection(make_float4(1, 1, 1, 1));
//
//	float center_ndc_x = center_eye.x / frustum.x;
//	center_ndc_x = (center_ndc_x + 1.0) * 0.5;
//	center_ndc_x = center_ndc_x * (float)width;
//	if (center_ndc_x < 0 || width <= center_ndc_x){
//		return;
//	}
//	float center_ndc_y = center_eye.y / frustum.y;
//	center_ndc_y = (center_ndc_y + 1.0) * 0.5;
//	center_ndc_y = center_ndc_y * (float)height;
//	if (center_ndc_y < 0 || width <= center_ndc_y){
//		return;
//	}
//
//	int _offset = (int)center_ndc_x + (int)center_ndc_y * width;
//	out[_offset] = center;
//}

__global__ void d_GaussianFilter_1D(float *out, float *rweight, int width, int height, int radius, float sigma_s, float sigma_r, bool vertical){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * width;
	if (offset >= width * height){
		return;
	}

	float center = tex2D(tex_depth, x, y);
	if (center == 1.0f){
		out[offset] = 1.0f;
		//rweight[offset] = backProjection(make_float4(0, 0, 0, 1)).x;
		return;
	}
	float4 center_eye = backProjection(projectionMatrix_3_2, projectionMatrix_2_2, invProjectionMatrix,
										make_float4(((float)x / (float)width) * 2.0f - 1.0f,
													((float)y / (float)height) * 2.0f - 1.0f,
													center * 2.0f - 1.0f,
													1.0f));

	float sum = 0.0f;
	float factor = 0.0f;
	float t = 0.0f;
	for (int i = -radius; i <= radius; ++i){
		int _x, _y;
		if (vertical){
			_x = x;
			_y = y + i;
		}
		else{
			_x = x + i;
			_y = y;
		}

		float sample = tex2D(tex_depth, _x, _y);
		if (sample == 1.0f){
			continue;
		}
		float4 sample_eye = backProjection(projectionMatrix_3_2, projectionMatrix_2_2, invProjectionMatrix,
											make_float4(((float)_x / (float)width) * 2.0f - 1.0f,
														((float)_y / (float)height) * 2.0f - 1.0f,
														sample * 2.0f - 1.0f,
														1.0f));
		float _i;
		if (vertical){
			_i = center_eye.y - sample_eye.y;
		}
		else{
			_i = center_eye.x - sample_eye.x;
		}
		//if (restorePos){
			factor = gaussian(_i, sigma_s) * euclideanLength(sample_eye.z, center_eye.z, sigma_r);
		//}
		//else{
		//	factor = gaussian(i, sigma_s) * euclideanLength(sample_eye.z, center_eye.z, sigma_r);
		//	//factor = d_Gaussian[i + radius] * euclideanLength(sample, center, sigma_r);
		//}

		t += factor * sample;
		sum += factor;
	}

	out[offset] = t / sum;
	//rweight[offset] = viewFrustumWidth;
}

__global__ void d_DOGFilter(float *out, int width, int height, int radius, float sigma1, float sigma2){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * width;
	if (offset >= width * height){
		return;
	}

	// Cross filtering
	float sum = 0.0f;
	for (int i = -radius; i <= radius; ++i){
		sum += d_Gaussian[i + radius] * tex2D(tex_depth, x + i, y);
		sum += d_Gaussian[i + radius] * tex2D(tex_depth, x, y + i);
	}
	out[offset] = sum / ((2 * radius + 1) + (2 * radius));

	// Normal filtering
	/*
	float sum = 0.0f;
	for (int i = -radius; i <= radius; ++i){
		for (int j = -radius; j <= radius; ++j){
			sum += (gaussian(i, j, sigma1) - gaussian(i, j, sigma2)) * tex2D(tex_depth, x + i, y + i);
		}
	}
	out[offset] = sum / ((2 * radius + 1) * (2 * radius + 1));
	*/

	// Box filtering
	/*
	float sum = 0.0f;
	for (int i = -1; i <= 1; ++i){
		for (int j = -1; j <= 1; ++j){
			sum += (gaussian(i, j, sigma1) - gaussian(i, j, sigma2)) * tex2D(tex_depth, x + i, y + i);
		}
	}
	out[offset] = sum / 9.0f;
	*/
}

__global__ void d_detectZeroCrossing(int width, int height){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * width;
	if (offset >= width * height){
		return;
	}

	float center = tex2D(tex_dogFiltered, x, y);
	unsigned char res = 0x00;
	if (center < 0){
		surf2Dwrite(res, surf_outline, x, y, hipBoundaryModeClamp);
		return;
	}
	float maxDiff = 0.0f;
	for (int i = -1; i <= 1; ++i){
		float sampleX = tex2D(tex_dogFiltered, x + i, y);
		if (center * sampleX < 0){
			float diff = fabs(center) + fabs(sampleX);
			if (diff > maxDiff){
				maxDiff = diff;
			}
		}

		float sampleY = tex2D(tex_dogFiltered, x, y + i);
		if (center * sampleY < 0){
			float diff = fabs(center) + fabs(sampleY);
			if (diff > maxDiff){
				maxDiff = diff;
			}
		}
	}

	if (maxDiff > 0.000001f){
	//if (maxDiff > 0.0f){
		res = 0xff;
	}
	surf2Dwrite(res, surf_outline, x, y, hipBoundaryModeClamp);
}

//__global__ void d_findMaxValue(float *out, int width, int height){
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//	int offset = x + y * width;
//	if (offset > width * height){
//		return;
//	}
//
//	__shared__ float sdata[16 * 16];
//	unsigned int tid = threadIdx.x + blockDim.x * threadIdx.y;
//	//sdata[tid] = tex2D(tex_depth, x, y);
//	sdata[tid] = getEyePos(tex2D(tex_depth, x, y));
//	__syncthreads();
//
//	for (unsigned int s = 1; s < blockDim.x * blockDim.y; s *= 2){
//		if (tid % (2 * s) == 0){
//			//float _max = fmaxf(sdata[tid], sdata[tid + s]);
//			//if (_max == 1.0f){
//			//	sdata[tid] = fminf(sdata[tid], sdata[tid + s]);
//			//}
//			//else{
//			//	sdata[tid] = _max;
//			//}
//
//			sdata[tid] = fmaxf(sdata[tid], sdata[tid + s]);
//		}
//		__syncthreads();
//	}
//
//	out[blockIdx.x + gridDim.x * blockIdx.y] = sdata[0];
//}
//
//__global__ void d_findMinValue(float *out, int width, int height){
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//	int offset = x + y * width;
//	if (offset > width * height){
//		return;
//	}
//
//	__shared__ float sdata[16 * 16];
//	unsigned int tid = threadIdx.x + blockDim.x * threadIdx.y;
//	//sdata[tid] = tex2D(tex_depth, x, y);
//	sdata[tid] = getEyePos(tex2D(tex_depth, x, y));
//	__syncthreads();
//
//	for (unsigned int s = 1; s < blockDim.x * blockDim.y; s *= 2){
//		if (tid % (2 * s) == 0){
//			sdata[tid] = fminf(sdata[tid], sdata[tid + s]);
//		}
//		__syncthreads();
//	}
//
//	out[blockIdx.x + gridDim.x * blockIdx.y] = sdata[0];
//}

__global__ void d_scaling(float *data, float min, float max, int bound){
	int offset = threadIdx.x + blockIdx.x * blockDim.x;
	if (offset > bound){
		return;
	}

	data[offset] = (data[offset] - min) / (max - min);
}

__global__ void test(float *canvas, int width, int height){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * width;
	if (offset > width * height){
		return;
	}

	if (offset < 2048){
		canvas[offset] = 0.0f;
	}
	else{
		canvas[offset] = tex2D(tex_depth, x, y);
	}
}

__global__ void avg(float *canvas1, float *canvas2, int width, int height){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * width;
	if (offset > width * height){
		return;
	}

	canvas1[offset] = (canvas1[offset] + canvas2[offset]) / 2;
}

BilateralFilter::BilateralFilter(int width, int height,
	GLuint depthTexID, int bf_radius, int bf_sigma_s, float bf_sigma_r, int bf_nIterations,
	GLuint outlineTexID, int dog_radius, int dog_sigma, float dog_similarity,
	GLuint thicknessTexID)
	: m_dev_canvas(NULL),
	m_width(width), m_height(height),
	m_grids1D((width * height + 256 - 1) / 256), m_threads1D(256), m_grids2D((width + 16 - 1) / 16, (height + 16 - 1) / 16), m_threads2D(16, 16),
	m_lowpassFilter(NULL)
{
	setBFParam(bf_radius, bf_sigma_s, bf_sigma_r, bf_nIterations);
	setDOGParam(dog_radius, dog_sigma, dog_similarity);

	checkCUDA(hipMalloc((void **)&m_dev_canvas, sizeof(float) * width * height),
		"Allocating device memory for filter.");

	checkCUDA(hipGraphicsGLRegisterImage(&m_graphicResources[0], depthTexID, GL_TEXTURE_2D, cudaGraphicsMapFlagsNone),
		"Registering graphic resources for depth texture.");
	checkCUDA(hipGraphicsGLRegisterImage(&m_graphicResources[1], outlineTexID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore),
		"Registering graphic resources for outline texture");
	checkCUDA(hipGraphicsGLRegisterImage(&m_graphicResources[2], thicknessTexID, GL_TEXTURE_2D, cudaGraphicsMapFlagsNone),
		"Registering graphic resources for thickness texture");

	m_lowpassFilter = LowpassFilter::create(width, height, 20.0f, true);
}

BilateralFilter::~BilateralFilter() {
	/* This will be needed if hipDeviceReset() would not be called */
	/*
	checkCUDA(hipFree(m_dev_canvas), "Deallocating device memory for filter.");

	checkCUDA(hipGraphicsUnregisterResource(m_graphicResources[0]),
		"Unregistering graphic resources for depth texture.");
	checkCUDA(hipGraphicsUnregisterResource(m_graphicResources[1]),
		"Unregistering graphic resources for outline texture.");
	checkCUDA(hipGraphicsUnregisterResource(resource[2]),
		"Unregistering graphic resources for thickness texture.");

	checkCUDA(hipFree(m_dev_complex), "Deallocating device memory for cufft");
	checkCUFFT(hipfftDestroy(m_plan_fwd), "Deallocating cufft foward plan");
	checkCUFFT(hipfftDestroy(m_plan_inv), "Deallocating cufft inverse plan");
	*/

	delete m_lowpassFilter;
}

void
BilateralFilter::filter(bool renderOutline){
	hipArray *arr_depth = NULL;
	hipArray *arr_outline = NULL;
	checkCUDA(hipGraphicsMapResources(2, m_graphicResources),
		"Mapping graphic resources.");
	checkCUDA(hipGraphicsSubResourceGetMappedArray(&arr_depth, m_graphicResources[0], 0, 0),
		"Getting mapped hipArray for depth texture.");
	checkCUDA(hipGraphicsSubResourceGetMappedArray(&arr_outline, m_graphicResources[1], 0, 0),
		"Getting mapped hipArray for outline texture.");

	hipChannelFormatDesc desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	checkCUDA(hipBindTextureToArray(tex_depth, arr_depth, desc),
		"Binding arr_depth to texture memory.");
	tex_depth.addressMode[0] = hipAddressModeMirror;
	tex_depth.addressMode[1] = hipAddressModeMirror;

	float *depth_original;
	hipMalloc((void **)&depth_original, sizeof(float) * m_width * m_height);
	hipMemcpyFromArray(depth_original, arr_depth, 0, 0, sizeof(float) * m_width * m_height, hipMemcpyDeviceToDevice);
	hipBindTexture2D(0, tex_depth_original, depth_original, desc, m_width, m_height, sizeof(float) * m_width);

	hipMemcpyToSymbol(HIP_SYMBOL(restorePos), &m_restorePos, sizeof(int));
	//std::cout << m_intensityRange << " " << m_bf_sigma_r << std::endl;

	//test << < m_grids2D, m_threads2D >> > (m_dev_canvas, m_width, m_height);
	//checkCUDA(hipGetLastError());
	//checkCUDA(hipMemcpyToArray(arr_depth, 0, 0, m_dev_canvas, sizeof(float) * m_width * m_height, hipMemcpyDeviceToDevice));

	//FIXME TEMP
	//float *dev_max;
	//hipMalloc(&dev_max, m_grids2D.x * m_grids2D.y * sizeof(float));
	//hipMemset(dev_max, 0, m_grids2D.x * m_grids2D.y * sizeof(float));
	//d_findMaxValue << <m_grids2D, m_threads2D >> >(dev_max, m_width, m_height);
	//std::vector<float> max(m_grids2D.x * m_grids2D.y);
	//hipMemcpy(max.data(), dev_max, m_grids2D.x * m_grids2D.y * sizeof(float), hipMemcpyDeviceToHost);
	//float maxv = -99999.0f;
	//for (auto &v : max){
	//	if (maxv < v){
	//		maxv = v;
	//	}
	//}
	//d_findMinValue << < m_grids2D, m_threads2D >> > (dev_max, m_width, m_height);
	//hipMemcpy(max.data(), dev_max, m_grids2D.x * m_grids2D.y * sizeof(float), hipMemcpyDeviceToHost);
	//float minv = 999999.0f;
	//for (auto &v : max){
	//	if (minv > v){
	//		minv = v;
	//	}
	//}
	//std::cout << maxv << " " << minv << std::endl;

	if (renderOutline){
		checkCUDA(hipBindTexture2D(0, tex_dogFiltered, m_dev_canvas, desc, m_width, m_height, sizeof(float) * m_width),
			"Binding dog filtered data to texture memory.");
		tex_dogFiltered.addressMode[0] = hipAddressModeMirror;
		tex_dogFiltered.addressMode[1] = hipAddressModeMirror;

		desc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
		checkCUDA(cudaBindSurfaceToArray(surf_outline, arr_outline, desc),
			"Binding arr_outline to surface memory.");

		checkCUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_Gaussian), m_dog_Gaussian.data(), sizeof(float) * m_dog_Gaussian.size()),
			"Memcpy to Gaussian symbol for dog filter.");
		d_DOGFilter << < m_grids2D, m_threads2D >> > (m_dev_canvas, m_width, m_height, m_dog_radius, static_cast<float>(m_dog_sigma), m_dog_similarity * static_cast<float>(m_dog_sigma));
		checkCUDA(hipGetLastError(), "Kernel function error : DOGFilter.");
		d_detectZeroCrossing << < m_grids2D, m_threads2D >> > (m_width, m_height);
		checkCUDA(hipGetLastError(), "Kernel function error : ExtractZeroCrossing.");

		checkCUDA(hipUnbindTexture(tex_dogFiltered), "Unbinding texture memory for dog filtered data.");
	}

	//hipMemcpyFromArray(m_dev_canvas, arr_depth, 0, 0, sizeof(float) * m_width * m_height, hipMemcpyDeviceToDevice);
	//d_scaling <<< m_grids1D, m_threads1D>>>(m_dev_canvas, minv, maxv, m_width * m_height);
	//hipMemcpyToArray(arr_depth, 0, 0, m_dev_canvas, sizeof(float) * m_width * m_height, hipMemcpyDeviceToDevice);



	//std::cout << "RESTORE POS : " << m_restorePos << " sigma r : " << m_bf_sigma_r << std::endl;

	float *dev_rweight;
	hipMalloc(&dev_rweight, m_width * m_height * sizeof(float));
	
	for (auto &Gaussian : m_bf_Gaussians){
		int radius = (Gaussian.size() >> 1);
		//checkCUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_Gaussian), Gaussian.data(), sizeof(float) * Gaussian.size()),
		//	"Memcpy to Gaussian symbol for bilateral filter.");

		d_GaussianFilter_1D << < m_grids2D, m_threads2D >> > (m_dev_canvas, dev_rweight, m_width, m_height, radius, m_bf_sigma_s, m_bf_sigma_r, true);
		checkCUDA(hipGetLastError(), "Kernel function error : GaussianFilter_Y.");
		checkCUDA(hipMemcpyToArray(arr_depth, 0, 0, m_dev_canvas, sizeof(float) * m_width * m_height, hipMemcpyDeviceToDevice),
			"Memcpy to arr_depth(texture memory)");

		d_GaussianFilter_1D << < m_grids2D, m_threads2D >> > (m_dev_canvas, dev_rweight, m_width, m_height, radius, m_bf_sigma_s, m_bf_sigma_r, false);
		checkCUDA(hipGetLastError(), "Kernel function error : GaussianFilter_X.");
		checkCUDA(hipMemcpyToArray(arr_depth, 0, 0, m_dev_canvas, sizeof(float) * m_width * m_height, hipMemcpyDeviceToDevice),
			"Memcpy to arr_depth(texture memory)");
	}

	//if (m_restorePos){
	//	checkCUDA(hipMemcpyToArray(arr_depth, 0, 0, depth_original, sizeof(float) * m_width * m_height, hipMemcpyDeviceToDevice),
	//		"Memcpy to arr_depth(texture memory)");

	//	for (auto &Gaussian : m_bf_Gaussians){
	//		int radius = (Gaussian.size() >> 1);

	//		d_GaussianFilter_1D << < m_grids2D, m_threads2D >> > (dev_rweight, dev_rweight, m_width, m_height, radius, m_bf_sigma_s, m_bf_sigma_r, false);
	//		checkCUDA(hipGetLastError(), "Kernel function error : GaussianFilter_Y.");
	//		checkCUDA(hipMemcpyToArray(arr_depth, 0, 0, dev_rweight, sizeof(float) * m_width * m_height, hipMemcpyDeviceToDevice),
	//			"Memcpy to arr_depth(texture memory)");

	//		d_GaussianFilter_1D << < m_grids2D, m_threads2D >> > (dev_rweight, dev_rweight, m_width, m_height, radius, m_bf_sigma_s, m_bf_sigma_r, true);
	//		checkCUDA(hipGetLastError(), "Kernel function error : GaussianFilter_X.");
	//		checkCUDA(hipMemcpyToArray(arr_depth, 0, 0, dev_rweight, sizeof(float) * m_width * m_height, hipMemcpyDeviceToDevice),
	//			"Memcpy to arr_depth(texture memory)");
	//	}

	//	avg << < m_grids2D, m_threads2D >> > (m_dev_canvas, dev_rweight, m_width, m_height);
	//	checkCUDA(hipMemcpyToArray(arr_depth, 0, 0, m_dev_canvas, sizeof(float) * m_width * m_height, hipMemcpyDeviceToDevice),
	//		"Memcpy to arr_depth(texture memory)");
	//}

	//if (m_restorePos){
	//	d_GaussianFilter << < m_grids2D, m_threads2D >> > (m_dev_canvas, m_width, m_height, 20, m_bf_sigma_s, m_bf_sigma_r);
	//	checkCUDA(hipGetLastError(), "Kernel function error : GaussianFilter_X.");
	//	checkCUDA(hipMemcpyToArray(arr_depth, 0, 0, m_dev_canvas, sizeof(float) * m_width * m_height, hipMemcpyDeviceToDevice),
	//		"Memcpy to arr_depth(texture memory)");
	//}

	//hipMemcpy(m_dev_canvas, std::vector<float>(m_width * m_height, 1.0f).data(), sizeof(float) *m_width * m_height, hipMemcpyHostToDevice);
	//ortho << < m_grids2D, m_threads2D >> > (m_dev_canvas, m_width, m_height);
	//	checkCUDA(hipGetLastError(), "Kernel function error : GaussianFilter_Y.");
	//	checkCUDA(hipMemcpyToArray(arr_depth, 0, 0, m_dev_canvas, sizeof(float) * m_width * m_height, hipMemcpyDeviceToDevice),
	//		"Memcpy to arr_depth(texture memory)");
	checkCUDA(hipUnbindTexture(tex_depth), "Unbinding texture memory for arr_depth");
	
	//std::vector<float> temp(m_width * m_height);
	//hipMemcpy(temp.data(), dev_rweight, sizeof(float) * m_width * m_height, hipMemcpyDeviceToHost);
	//float min = 0, max = 0;
	//for (int i = 0; i < m_width * m_height; ++i){
	//	if (temp[i] < min){
	//		min = temp[i];
	//	}
	//	if (temp[i] > max){
	//		max = temp[i];
	//	}
	//}
	//cout << "min : " << min << ", max : " << max << endl;
	hipFree(dev_rweight);

	hipUnbindTexture(tex_depth_original);
	hipFree(depth_original);

	checkCUDA(hipGraphicsUnmapResources(2, m_graphicResources),
		"Error : Unmapping graphic resources");
}

void
BilateralFilter::setBFParam(int radius, int sigma_s, float sigma_r, int nIterations) {
	m_bf_radius = radius;
	m_bf_sigma_s = sigma_s;
	m_bf_sigma_r = sigma_r;
	m_bf_nIterations = nIterations;

	m_bf_Gaussians.clear();
	for (int i = 0; i < nIterations; ++i){
		std::vector<float> Gaussian;
		for (int i = 0; i < 2 * radius + 1; ++i){
			Gaussian.push_back(gaussian(i - radius, sigma_s));
		}
		m_bf_Gaussians.push_back(Gaussian);
		radius >>= 1;
		if (radius == 0){
			break;
		}
	}
}

void
BilateralFilter::setDOGParam(int radius, int sigma, float similarity){
	m_dog_radius = radius;
	m_dog_sigma = sigma;
	m_dog_similarity = similarity;

	m_dog_Gaussian.clear();
	for (int i = 0; i < 2 * radius + 1; ++i){
		m_dog_Gaussian.push_back(gaussian(i - radius, sigma) - gaussian(i - radius, similarity * sigma));
	}
}

void
BilateralFilter::setDepthTexture(GLuint depthTexID){
	checkCUDA(hipGraphicsUnregisterResource(m_graphicResources[0]),
		"Error : Unregistering graphic resource for depth texture.");
	checkCUDA(hipGraphicsGLRegisterImage(&m_graphicResources[0], depthTexID, GL_TEXTURE_2D, cudaGraphicsMapFlagsNone),
		"Error : Registering graphic resource for depth texture.");
}

void
BilateralFilter::setOutlineTexture(GLuint outlineTexID){
	checkCUDA(hipGraphicsUnregisterResource(m_graphicResources[1]),
		"Error : Unregistering graphic resource for outline texture.");
	checkCUDA(hipGraphicsGLRegisterImage(&m_graphicResources[1], outlineTexID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore),
		"Error : Registering graphic resource for outline texture.");
}

void
BilateralFilter::setThicknessTexture(GLuint thicknessTexID) {
	checkCUDA(hipGraphicsUnregisterResource(m_graphicResources[2]),
		"Error : Unregistering graphic resource for thickness texture.");
	checkCUDA(hipGraphicsGLRegisterImage(&m_graphicResources[2], thicknessTexID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore),
		"Error : Registering graphic resource for thickness texture.");
}

void
BilateralFilter::filterThickness() {
	hipArray *arr_tex = NULL;
	checkCUDA(hipGraphicsMapResources(1, &m_graphicResources[2]),
		"Mapping graphic resources.");
	checkCUDA(hipGraphicsSubResourceGetMappedArray(&arr_tex, m_graphicResources[2], 0, 0),
		"Getting mapped hipArray for thickness texture.");

	checkCUDA(hipMemcpyFromArray(m_dev_canvas, arr_tex, 0, 0, m_width * m_height * sizeof(float), hipMemcpyDeviceToDevice),
		"Memcpy from arr_tex to device memory.");
	
	m_lowpassFilter->run(m_dev_canvas);

	checkCUDA(hipMemcpyToArray(arr_tex, 0, 0, m_dev_canvas, m_width * m_height* sizeof(float), hipMemcpyDeviceToDevice),
		"Memcpy from device memory to arr_tex");

	checkCUDA(hipGraphicsUnmapResources(1, &m_graphicResources[2]),
		"Error : Unmapping graphic resources");
} 

void BilateralFilter::setProjectionMatrix(glm::mat4 matrix){
	checkCUDA(hipMemcpyToSymbol(HIP_SYMBOL(projectionMatrix_3_2), &(matrix[3][2]), sizeof(float)),
		"Memcpy to projmat32 for bilateral filter.");
	checkCUDA(hipMemcpyToSymbol(HIP_SYMBOL(projectionMatrix_2_2), &(matrix[2][2]), sizeof(float)),
		"Memcpy to projmat22 for bilateral filter.");

	glm::mat4 inv = glm::inverse(matrix);
	checkCUDA(hipMemcpyToSymbol(HIP_SYMBOL(invProjectionMatrix), &inv, sizeof(glm::mat4)),
		"Memcpy to invprojmat for bilateral filter.");

	m_P = matrix;
}